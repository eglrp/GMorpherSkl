#include "hip/hip_runtime.h"
#include "../kernels_include/kernel_ICP.h"
#include <stdio.h>
#include <float.h>
//#include <cutil_math.h>



texture<int, 1, hipReadModeElementType>    XNboundsTex;

__global__ void CUDA_EStep(const int     numTargets,
                            const int     numPatches,
                            const float   sigma,
                            const float   EOutlier,
                            const float   dotThresh,
                            const float3* XNt,
                            const float3* XN,
                            float*  weights,
                            int*    targets )
{
	unsigned int vit = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if( vit >= numTargets ) return;

	float* w_line = weights + (numPatches+1)*vit;
	int*   t_line = targets + (numPatches+1)*vit;

	float sumWeights = 0.0f;
	float  half_sigmainv2 = 0.5f/(sigma*sigma);

	const float3 Xt = XNt[2*vit];
	const float3 Nt = XNt[2*vit+1];

	const float3* xn_ptr = XN;
	for( int pi=0; pi<numPatches; ++pi)
	{
		float minVal = FLT_MAX;
		int minIdx   = 0;
		const float3* end_ptr   = XN + 2*tex1Dfetch(XNboundsTex, pi+1 );

		while( xn_ptr != end_ptr ) {
			float3 delta = make_float3(xn_ptr[0].x - Xt.x, xn_ptr[0].y - Xt.y, xn_ptr[0].z - Xt.z);
			float  dist2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
			float  dotN = xn_ptr[1].x*Nt.x + xn_ptr[1].y*Nt.y + xn_ptr[1].z*Nt.z;
			if( (dotN > dotThresh) && (dist2 < minVal ) ){
				minVal = dist2;
				minIdx = xn_ptr - XN;
			}
			xn_ptr +=2;
		}

		if( minVal == FLT_MAX ) w_line[pi] = 0;
		else {
			float wi = expf(-minVal*half_sigmainv2);
			sumWeights+= wi;
			w_line[pi] = wi;
		}
		t_line[pi] = minIdx; // since we have the indices multiplied by 2
	}

	// -----------------------------
	// 2 - normalize
	w_line[numPatches]    = EOutlier*sigma*sigma;
	t_line[numPatches]    = -1;
	sumWeights += EOutlier*sigma*sigma;

	for(int pi=0;pi<numPatches+1;++pi) w_line[pi] /= sumWeights;
}




void runKernel_ICP_EStep( int         numPatches,
                          int         numTargets,
                          float       sigma,
                          float       dotThresh,
                          float       EOutlier,
                          const void* XNt_device,
                          const void* XN_device,
                          const void* XN_bounds_device,
                          void*       weights_device,
                          void*       targets_device )
{
	// 1 - find the correct number of blocks
	int numBlocks   = (numTargets / CUDA_ICP_ESTEP_TPB);
	if( numTargets % CUDA_ICP_ESTEP_TPB ) numBlocks++;


	hipEvent_t startEvt, stopEvt;
	hipEventCreate(&startEvt);
	hipEventCreate(&stopEvt);
	hipEventRecord(startEvt,0);


	// ###################################
	hipChannelFormatDesc channeldesc = hipCreateChannelDesc<int>();
	hipBindTexture(0, XNboundsTex, XN_bounds_device, channeldesc,  (numPatches+1)*sizeof(int));

	CUDA_EStep<<<numBlocks, CUDA_ICP_ESTEP_TPB>>>( numTargets,
	                                               numPatches,
	                                               sigma,
	                                               EOutlier,
	                                               dotThresh,
	                                               (const float3*)XNt_device,
	                                               (const float3*)XN_device,
	                                               (float*) weights_device,
	                                               (int*) targets_device );

	hipUnbindTexture( XNboundsTex);
	// ###################################

	hipEventRecord(stopEvt,0);
	hipDeviceSynchronize();
	float ms = 0.0;
	hipEventElapsedTime (&ms, startEvt, stopEvt);
	hipEventDestroy(startEvt);
	hipEventDestroy(stopEvt);

	printf("CUDA -- spent %f ms in the kernel %s \n", ms , __FUNCTION__);

	hipError_t status = hipGetLastError();
	if(status != hipSuccess){
		printf("(E) CUDA error: %s\n %s\n", __FUNCTION__, hipGetErrorString(status));
	}
}
