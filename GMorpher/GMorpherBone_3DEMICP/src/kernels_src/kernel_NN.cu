#include "hip/hip_runtime.h"
#include "../kernels_include/kernel_NN.h"
#include <stdio.h>
#include <float.h>
#include <cutil_math.h>



texture<int, 1, hipReadModeElementType>    XNsmoothboundsTex;

__global__ void CUDA_NN(const int     numTargets,
						const int     numPatches,
						const float   sigma,
						const float   EOutlier,
						const float   dotThresh,
						const float3* XNt,
						const float3* XN_smooth,
						float*  weights,
						int*    targets )
{
	uint vit = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if( vit >= numTargets ) return;

	float* w_line = weights + (numPatches+1)*vit;
	int*   t_line = targets + (numPatches+1)*vit;

	float sumWeights = 0.0f;
	float  half_sigmainv2 = 0.5f/(sigma*sigma);

	const float3 Xt = XNt[2*vit];
	const float3 Nt = XNt[2*vit+1];

	const float3* xn_ptr = XN_smooth;

	w_line[numPatches] = 1;	
	for( int pi=0; pi<numPatches; ++pi)
	{
		float minVal = FLT_MAX;
		const float3* minIdx   = xn_ptr;
		const float3* const beg_ptr = xn_ptr;
		const float3* const end_ptr = XN_smooth + 2*tex1Dfetch(XNsmoothboundsTex, pi+1 );

		while( xn_ptr != end_ptr ) {
			float3 delta = xn_ptr[0] - Xt;
			float  dist2 = dot(delta,delta);
			float  dotN  = dot( xn_ptr[1], Nt );
			if( (dotN > dotThresh) && (dist2 < minVal ) ){
				minVal = dist2;
				minIdx = xn_ptr;
			}
			xn_ptr +=2;
		}

		if( minVal == FLT_MAX ) w_line[pi] = 0;
		else {
			float wi = __expf(-minVal*half_sigmainv2);
			sumWeights+= wi;
			w_line[pi] = wi;
			w_line[numPatches] *= (1 - wi);
		}
		t_line[pi] = minIdx - beg_ptr; // since we have the indices multiplied by 2
	}

	//// -----------------------------
	//// 2 - normalize
	w_line[numPatches]    = EOutlier*sigma*sigma;
	t_line[numPatches]    = -1;
	sumWeights += w_line[numPatches];

	for(int pi=0;pi<numPatches+1;++pi) w_line[pi] /= sumWeights;
}

void runKernel_ICPPred_EStep( int         numPatches,
                              int         numTargets,
                              float       sigma,
                              float       dotThresh,
                              float       EOutlier,
                              const void* XNt_device,
                              const void* XN_smooth_device,
                              const void* XN_smooth_bounds_device,
                              void*       weights_device,
                              void*       targets_device)
{
	// 1 - find the correct number of blocks
	int numBlocks   = (numTargets / CUDA_NN_TPB);
	if( numTargets % CUDA_NN_TPB ) numBlocks++;


	//printf("launching kernel with %d blocks \n", numBlocks );
	hipEvent_t startEvt, stopEvt;
	hipEventCreate(&startEvt);
	hipEventCreate(&stopEvt);
	hipEventRecord(startEvt,0);


	// ###################################
	hipChannelFormatDesc channeldesc = hipCreateChannelDesc<int>();
	hipBindTexture(0, XNsmoothboundsTex, XN_smooth_bounds_device, channeldesc,  (numPatches+1)*sizeof(int));

	CUDA_PredEStep<<<numBlocks, CUDA_NN_TPB>>>( numTargets,
	                                                       numPatches,
	                                                       sigma,
	                                                       EOutlier,
	                                                       dotThresh,
	                                                       (const float3*)XNt_device,
	                                                       (const float3*)XN_smooth_device,
	                                                       (float*)       weights_device,
	                                                       (int*)         targets_device);

	
	hipUnbindTexture( XNsmoothboundsTex);
	// ###################################

	hipEventRecord(stopEvt,0);
	hipDeviceSynchronize();
	float ms = 0.0;
	hipEventElapsedTime (&ms, startEvt, stopEvt);
	hipEventDestroy(startEvt);
	hipEventDestroy(stopEvt);

	//printf("CUDA -- spent %f ms in the kernel %s \n", ms , __FUNCTION__);

	{
		hipError_t status = hipGetLastError();
		if(status != hipSuccess) printf("(E) CUDA error: %s:%d\n %s\n", __FUNCTION__, __LINE__, hipGetErrorString(status));
	}

}
